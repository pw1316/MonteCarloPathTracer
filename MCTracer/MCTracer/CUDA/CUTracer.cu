#include "hip/hip_runtime.h"
#include "CUTracer.h"

#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <thrust/device_vector.h>

#include <vector>

#include "Framework/Geometry.h"

namespace PW
{
	namespace Tracer
	{
		const PWuint IMG_WIDTH = 800;
		const PWuint IMG_HEIGHT = 600;
		const PWuint NUM_SAMPLES = 4;

		__device__ PWVector3f* vertexBuffer;
		__device__ PWuint nVertexBuffer;
		__device__ PWVector3f* normalBuffer;
		__device__ PWuint nNormalBuffer;
		__device__ Geometry::Triangle* triangleBuffer;
		__device__ PWuint nTriangleBuffer;
		__device__ Geometry::Geometry* geometryBuffer;
		__device__ PWuint nGeometryBuffer;

		__global__ void rayTraceKernel(PWVector4f *c, PWuint seedOffset)
		{
			extern __shared__ PWVector4f sampleBuffer[];
			PWuint x = blockIdx.x;
			PWuint y = blockIdx.y;
			PWuint width = gridDim.x;
			PWuint height = gridDim.y;
			PWuint sampleId = threadIdx.x;
			/* Init RNG */
			hiprandState stateRNG;
			hiprand_init(sampleId + seedOffset, 0, 0, &stateRNG);
			/* Camera Params inline */
			PWVector3f camEye(0, 5, 17);
			PWVector3f camDir(0, 0, -1);
			PWVector3f camUp(0, 1, 0);
			PWVector3f camRight(1, 0, 0);
			/* Project Params inline */
			PWfloat projFOV = 60; // degree
			PWfloat projNear = 1;
			PWfloat projFar = 25;
			PWfloat mathPI = 3.14159265359f;
			
			/* Reproject */
			PWVector3f initRayDir;
			initRayDir.x = (2.0 * x / width + 1) * tan(projFOV * mathPI / 360);
			initRayDir.y = (2.0 * y / width + 1.0 * height / width) * tan(projFOV * mathPI / 360);
			initRayDir.z = -1;
			/* MC Sampling */
			/// TODO
			sampleBuffer[threadIdx.x].x = initRayDir.x;
			sampleBuffer[threadIdx.x].y = initRayDir.y;
			sampleBuffer[threadIdx.x].z = initRayDir.z;
			sampleBuffer[threadIdx.x].w = 0;
			__syncthreads();
			/* Reduce SUM test */
			for (PWuint s = blockDim.x / 2; s > 0; s >>= 1)
			{
				if (threadIdx.x < s)
				{
					sampleBuffer[threadIdx.x].x += sampleBuffer[threadIdx.x + s].x;
					sampleBuffer[threadIdx.x].y += sampleBuffer[threadIdx.x + s].y;
					sampleBuffer[threadIdx.x].z += sampleBuffer[threadIdx.x + s].z;
					sampleBuffer[threadIdx.x].w += sampleBuffer[threadIdx.x + s].w;
				}
				__syncthreads();
			}
			if (threadIdx.x == 0)
			{
				c[y * width + x].x = sampleBuffer[0].x / blockDim.x;
				c[y * width + x].y = sampleBuffer[0].y / blockDim.x;
				c[y * width + x].z = sampleBuffer[0].z / blockDim.x;
				c[y * width + x].w = sampleBuffer[0].w / blockDim.x;
			}
		}

		hipError_t RenderScene1(PW::FileReader::ObjModel *model)
		{
			hipError_t cudaStatus;
			cudaStatus = hipSetDevice(0);
			if (cudaStatus != hipSuccess)
			{
				return cudaStatus;
			}

			PWuint deviceVertexBufferNum = model->m_vertices.size();
			hipMemcpyToSymbol(HIP_SYMBOL(nVertexBuffer), &deviceVertexBufferNum, sizeof(PWuint));
			void* deviceVertexBufferAddr = nullptr;
			hipMalloc((void**)&deviceVertexBufferAddr, sizeof(PWVector3f) * deviceVertexBufferNum);
			hipMemcpyToSymbol(HIP_SYMBOL(vertexBuffer), &deviceVertexBufferAddr, sizeof(void*));
			std::vector<PWVector3f> hostVertexBuffer(deviceVertexBufferNum);
			for (PWuint i = 0; i < deviceVertexBufferNum; ++i)
			{
				hostVertexBuffer[i].x = model->m_vertices[i].getX();
				hostVertexBuffer[i].y = model->m_vertices[i].getY();
				hostVertexBuffer[i].z = model->m_vertices[i].getZ();
			}
			hipMemcpy(deviceVertexBufferAddr, &hostVertexBuffer[0], sizeof(PWVector3f) * deviceVertexBufferNum, hipMemcpyHostToDevice);

			PWuint deviceNormalBufferNum = model->m_normals.size();
			hipMemcpyToSymbol(HIP_SYMBOL(nNormalBuffer), &deviceNormalBufferNum, sizeof(PWuint));
			void* deviceNormalBufferAddr = nullptr;
			hipMalloc((void**)&deviceNormalBufferAddr, sizeof(PWVector3f) * deviceNormalBufferNum);
			hipMemcpyToSymbol(HIP_SYMBOL(normalBuffer), &deviceNormalBufferAddr, sizeof(void*));
			std::vector<PWVector3f> hostNormalBuffer(deviceNormalBufferNum);
			for (PWuint i = 0; i < deviceNormalBufferNum; ++i)
			{
				hostNormalBuffer[i].x = model->m_normals[i].getX();
				hostNormalBuffer[i].y = model->m_normals[i].getY();
				hostNormalBuffer[i].z = model->m_normals[i].getZ();
			}
			hipMemcpy(deviceVertexBufferAddr, &hostNormalBuffer[0], sizeof(PWVector3f) * deviceNormalBufferNum, hipMemcpyHostToDevice);

			PWuint deviceTriangleBufferNum = model->m_triangles.size();
			hipMemcpyToSymbol(HIP_SYMBOL(nTriangleBuffer), &deviceTriangleBufferNum, sizeof(PWuint));
			void* deviceTriangleBufferAddr = nullptr;
			hipMalloc((void**)&deviceTriangleBufferAddr, sizeof(Geometry::Triangle) * deviceTriangleBufferNum);
			hipMemcpyToSymbol(HIP_SYMBOL(triangleBuffer), &deviceTriangleBufferAddr, sizeof(void*));
			std::vector<Geometry::Triangle> hostTriangleBuffer(deviceTriangleBufferNum);
			for (PWuint i = 0; i < deviceTriangleBufferNum; ++i)
			{
				hostTriangleBuffer[i].v[0] = model->m_triangles[i].m_vertexIndex[0];
				hostTriangleBuffer[i].v[1] = model->m_triangles[i].m_vertexIndex[1];
				hostTriangleBuffer[i].v[2] = model->m_triangles[i].m_vertexIndex[2];
				hostTriangleBuffer[i].n[0] = model->m_triangles[i].m_normalIndex[0];
				hostTriangleBuffer[i].n[1] = model->m_triangles[i].m_normalIndex[1];
				hostTriangleBuffer[i].n[2] = model->m_triangles[i].m_normalIndex[2];
			}
			hipMemcpy(deviceTriangleBufferAddr, &hostTriangleBuffer[0], sizeof(Geometry::Triangle) * deviceTriangleBufferNum, hipMemcpyHostToDevice);

			PWuint deviceGeometryBufferNum = 0;
			for (auto &group : model->m_groups)
			{
				if (group.second.m_triangleIndices.size() != 0)
				{
					deviceGeometryBufferNum += 1;
				}
			}
			hipMemcpyToSymbol(HIP_SYMBOL(nGeometryBuffer), &deviceGeometryBufferNum, sizeof(PWuint));
			void* deviceGeometryBufferAddr = nullptr;
			hipMalloc((void**)&deviceGeometryBufferAddr, sizeof(Geometry::Geometry) * deviceGeometryBufferNum);
			hipMemcpyToSymbol(HIP_SYMBOL(geometryBuffer), &deviceGeometryBufferAddr, sizeof(void*));
			std::vector<Geometry::Geometry> hostGeometryBuffer(deviceGeometryBufferNum);
			PWuint gIndex = 0;
			for (auto &group : model->m_groups)
			{
				if (group.second.m_triangleIndices.size() == 0)
				{
					continue;
				}
				auto &triB = model->m_triangles;
				auto &matB = model->m_materials;
				hostGeometryBuffer[gIndex].startIndex = group.second.m_triangleIndices[0];
				hostGeometryBuffer[gIndex].numTriangles = group.second.m_triangleIndices.size();
				auto &mat = matB[triB[group.second.m_triangleIndices[0]].materialIndex];
				hostGeometryBuffer[gIndex].material.Ka = PWVector3f(mat.Ka.getX(), mat.Ka.getY(), mat.Ka.getZ());
				hostGeometryBuffer[gIndex].material.Kd = PWVector3f(mat.Kd.getX(), mat.Kd.getY(), mat.Kd.getZ());
				hostGeometryBuffer[gIndex].material.Ks = PWVector3f(mat.Ks.getX(), mat.Ks.getY(), mat.Ks.getZ());
				hostGeometryBuffer[gIndex].material.Ns = mat.Ns;
				hostGeometryBuffer[gIndex].material.Tr = mat.Tr;
				hostGeometryBuffer[gIndex].material.Ni = mat.Ni;
				gIndex += 1;
			}
			hipMemcpy(deviceGeometryBufferAddr, &hostGeometryBuffer[0], sizeof(Geometry::Geometry) * deviceGeometryBufferNum, hipMemcpyHostToDevice);

			PWVector4f *color = nullptr;
			cudaStatus = hipMalloc((void**)&color, IMG_WIDTH * IMG_HEIGHT * sizeof(PWVector4f));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				hipFree(color);
				return cudaStatus;
			}
			// Launch a kernel on the GPU with one thread for each element.
			dim3 gridSize(IMG_WIDTH, IMG_HEIGHT);
			dim3 blockSize(NUM_SAMPLES);
			rayTraceKernel << <gridSize, blockSize, NUM_SAMPLES * sizeof(PWVector4f) >> > (color, 0);
			hipDeviceSynchronize();
			PWVector4f *hostcolor = new PWVector4f[IMG_WIDTH * IMG_HEIGHT]; // Width*Height
			hipMemcpy(hostcolor, color, IMG_WIDTH * IMG_HEIGHT * sizeof(PWVector4f), hipMemcpyDeviceToHost);

			return cudaStatus;
		}
	}
}
