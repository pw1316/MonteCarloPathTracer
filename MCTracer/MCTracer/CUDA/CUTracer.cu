#include "hip/hip_runtime.h"
#include "CUTracer.h"
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <vector>

#include "Framework/Geometry.h"

namespace PW
{
    namespace Tracer
    {
        __device__ PWVector3f* vertexBuffer;
        __device__ PWuint nVertexBuffer;
        __device__ PWVector3f* normalBuffer;
        __device__ PWuint nNormalBuffer;
        __device__ Geometry::Triangle* triangleBuffer;
        __device__ PWuint nTriangleBuffer;
        __device__ Geometry::Geometry* geometryBuffer;
        __device__ PWuint nGeometryBuffer;

        __device__ int* a;
        __device__ int* b;

        __global__ void addKernel(PWVector3f *c)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            c[i] = geometryBuffer[i].material.Kd;
        }

        __global__ void kernel(int *c)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            c[i] = a[i] + b[i];
        }

        hipError_t RenderScene1(PW::FileReader::ObjModel *model)
        {
            hipError_t cudaStatus;
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
            {
                return cudaStatus;
            }

            PWuint deviceVertexBufferNum = model->m_vertices.size();
            hipMemcpyToSymbol(HIP_SYMBOL(nVertexBuffer), &deviceVertexBufferNum, sizeof(PWuint));
            void* deviceVertexBufferAddr = nullptr;
            hipMalloc((void**)&deviceVertexBufferAddr, sizeof(PWVector3f) * deviceVertexBufferNum);
            hipMemcpyToSymbol(HIP_SYMBOL(vertexBuffer), &deviceVertexBufferAddr, sizeof(void*));
            std::vector<PWVector3f> hostVertexBuffer(deviceVertexBufferNum);
            for (PWuint i = 0; i < deviceVertexBufferNum; ++i)
            {
                hostVertexBuffer[i].x = model->m_vertices[i].getX();
                hostVertexBuffer[i].y = model->m_vertices[i].getY();
                hostVertexBuffer[i].z = model->m_vertices[i].getZ();
            }
            hipMemcpy(deviceVertexBufferAddr, &hostVertexBuffer[0], sizeof(PWVector3f) * deviceVertexBufferNum, hipMemcpyHostToDevice);

            PWuint deviceNormalBufferNum = model->m_normals.size();
            hipMemcpyToSymbol(HIP_SYMBOL(nNormalBuffer), &deviceNormalBufferNum, sizeof(PWuint));
            void* deviceNormalBufferAddr = nullptr;
            hipMalloc((void**)&deviceNormalBufferAddr, sizeof(PWVector3f) * deviceNormalBufferNum);
            hipMemcpyToSymbol(HIP_SYMBOL(normalBuffer), &deviceNormalBufferAddr, sizeof(void*));
            std::vector<PWVector3f> hostNormalBuffer(deviceNormalBufferNum);
            for (PWuint i = 0; i < deviceNormalBufferNum; ++i)
            {
                hostNormalBuffer[i].x = model->m_normals[i].getX();
                hostNormalBuffer[i].y = model->m_normals[i].getY();
                hostNormalBuffer[i].z = model->m_normals[i].getZ();
            }
            hipMemcpy(deviceVertexBufferAddr, &hostNormalBuffer[0], sizeof(PWVector3f) * deviceNormalBufferNum, hipMemcpyHostToDevice);

            PWuint deviceTriangleBufferNum = model->m_triangles.size();
            hipMemcpyToSymbol(HIP_SYMBOL(nTriangleBuffer), &deviceTriangleBufferNum, sizeof(PWuint));
            void* deviceTriangleBufferAddr = nullptr;
            hipMalloc((void**)&deviceTriangleBufferAddr, sizeof(Geometry::Triangle) * deviceTriangleBufferNum);
            hipMemcpyToSymbol(HIP_SYMBOL(triangleBuffer), &deviceTriangleBufferAddr, sizeof(void*));
            std::vector<Geometry::Triangle> hostTriangleBuffer(deviceTriangleBufferNum);
            for (PWuint i = 0; i < deviceTriangleBufferNum; ++i)
            {
                hostTriangleBuffer[i].v[0] = model->m_triangles[i].m_vertexIndex[0];
                hostTriangleBuffer[i].v[1] = model->m_triangles[i].m_vertexIndex[1];
                hostTriangleBuffer[i].v[2] = model->m_triangles[i].m_vertexIndex[2];
                hostTriangleBuffer[i].n[0] = model->m_triangles[i].m_normalIndex[0];
                hostTriangleBuffer[i].n[1] = model->m_triangles[i].m_normalIndex[1];
                hostTriangleBuffer[i].n[2] = model->m_triangles[i].m_normalIndex[2];
            }
            hipMemcpy(deviceTriangleBufferAddr, &hostTriangleBuffer[0], sizeof(Geometry::Triangle) * deviceTriangleBufferNum, hipMemcpyHostToDevice);

            PWuint deviceGeometryBufferNum = 0;
            for (auto &group : model->m_groups)
            {
                if (group.second.m_triangleIndices.size() != 0)
                {
                    deviceGeometryBufferNum += 1;
                }
            }
            hipMemcpyToSymbol(HIP_SYMBOL(nGeometryBuffer), &deviceGeometryBufferNum, sizeof(PWuint));
            void* deviceGeometryBufferAddr = nullptr;
            hipMalloc((void**)&deviceGeometryBufferAddr, sizeof(Geometry::Geometry) * deviceGeometryBufferNum);
            hipMemcpyToSymbol(HIP_SYMBOL(geometryBuffer), &deviceGeometryBufferAddr, sizeof(void*));
            std::vector<Geometry::Geometry> hostGeometryBuffer(deviceGeometryBufferNum);
            PWuint gIndex = 0;
            for (auto &group : model->m_groups)
            {
                if (group.second.m_triangleIndices.size() == 0)
                {
                    continue;
                }
                auto &triB = model->m_triangles;
                auto &matB = model->m_materials;
                hostGeometryBuffer[gIndex].startIndex = group.second.m_triangleIndices[0];
                hostGeometryBuffer[gIndex].numTriangles = group.second.m_triangleIndices.size();
                auto &mat = matB[triB[group.second.m_triangleIndices[0]].materialIndex];
                hostGeometryBuffer[gIndex].material.Ka = PWVector3f(mat.Ka.getX(), mat.Ka.getY(), mat.Ka.getZ());
                hostGeometryBuffer[gIndex].material.Kd = PWVector3f(mat.Kd.getX(), mat.Kd.getY(), mat.Kd.getZ());
                hostGeometryBuffer[gIndex].material.Ks = PWVector3f(mat.Ks.getX(), mat.Ks.getY(), mat.Ks.getZ());
                hostGeometryBuffer[gIndex].material.Ns = mat.Ns;
                hostGeometryBuffer[gIndex].material.Tr = mat.Tr;
                hostGeometryBuffer[gIndex].material.Ni = mat.Ni;
                gIndex += 1;
            }
            hipMemcpy(deviceGeometryBufferAddr, &hostGeometryBuffer[0], sizeof(Geometry::Geometry) * deviceGeometryBufferNum, hipMemcpyHostToDevice);

            PWVector3f *color = nullptr;
            cudaStatus = hipMalloc((void**)&color, 4 * sizeof(PWVector3f));
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMalloc failed!");
                hipFree(color);
                return cudaStatus;
            }
            // Launch a kernel on the GPU with one thread for each element.
            addKernel << <1, 4 >> > (color);
            PWVector4f *hostcolor = new PWVector4f(800 * 600 * 1024); // Width*Height*Samples
            hipMemcpy(hostcolor, color, 4 * sizeof(PWVector3f), hipMemcpyDeviceToHost);

            void* devAddr = nullptr;
            hipGetSymbolAddress(&devAddr, HIP_SYMBOL(a));
            std::vector<int> hosta(4);
            for (int i = 0; i < 4; i++) hosta[i] = i;
            hipMalloc((void**)&devAddr, sizeof(int) * 4);
            hipMemcpy(a, &hosta[0], sizeof(int) * 4, hipMemcpyHostToDevice);
            std::vector<int> hostb(4);
            for (int i = 0; i < 4; i++) hostb[i] = 10 * i;
            hipMalloc((void**)&b, sizeof(int) * 4);
            hipMemcpy(b, &hostb[0], sizeof(int) * 4, hipMemcpyHostToDevice);
            int *d_c = nullptr;
            hipMalloc((void**)d_c, sizeof(int) * 4);
            kernel << <1, 4 >> > (d_c);
            int hostc[4];

            hipDeviceSynchronize();
            hipMemcpy(hostc, d_c, sizeof(int) * 4, hipMemcpyDeviceToHost);
            //
            //    // Check for any errors launching the kernel
            //    cudaStatus = hipGetLastError();
            //    if (cudaStatus != hipSuccess) {
            //        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            //        goto Error;
            //    }
            //
            //    // hipDeviceSynchronize waits for the kernel to finish, and returns
            //    // any errors encountered during the launch.
            //    cudaStatus = hipDeviceSynchronize();
            //    if (cudaStatus != hipSuccess) {
            //        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            //        goto Error;
            //    }
            //
            //    // Copy output vector from GPU buffer to host memory.
            //    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
            //    if (cudaStatus != hipSuccess) {
            //        fprintf(stderr, "hipMemcpy failed!");
            //        goto Error;
            //    }
            //
            //Error:
            //    hipFree(dev_c);
            //    hipFree(dev_a);
            //    hipFree(dev_b);

            return cudaStatus;
        }
    }
}
